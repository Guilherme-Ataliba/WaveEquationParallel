#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "auxilary.c"

/*
    This is the shared memory version
    - No halo regions are required since they can be accessed through the global memory
*/

#define Nx 800
#define Ny 800
#define T 10
#define a 10.0
#define b 10.0
#define Cx 0.5
#define Cy 0.5
#define frames 20

__device__ double f(int i, int j, double t) {
    if (i < (Nx / 2 + Nx / 100) && i > (Nx / 2 - Nx / 100)) {
        if (j < (Ny / 2 + Ny / 100) && j > (Ny / 2 - Ny / 100)) {
            return 30 * sin(4.7123889 * t);
        }
    }
    return 0.0;
}

__global__ void step_zero_kernel(double* unm2) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < Nx && j < Ny) {
        int idx = j * Nx + i;
        unm2[idx] = 0; // Initial condition
    }
}

__global__ void step_one_kernel(double* unm1, double* unm2,double Cx2, double Cy2, double dt2) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i > 0 && i < Nx - 1 && j > 0 && j < Ny - 1) {
        int idx = j * Nx + i;
        unm1[idx] = unm2[idx]
                    + 0.5 * Cx2 * (unm2[idx + 1] - 2 * unm2[idx] + unm2[idx - 1])
                    + 0.5 * Cy2 * (unm2[idx + Nx] - 2 * unm2[idx] + unm2[idx - Nx])
                    + 0.5 * dt2 * f(i, j, 1);
    }
}

__global__ void next_step_kernel(double* u, double* unm1, double* unm2, double t, double Cx2, double Cy2, double dt2) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i > 0 && i < Nx - 1 && j > 0 && j < Ny - 1) {
        int idx = j * Nx + i;
        u[idx] = -unm2[idx] + 2 * unm1[idx]
                 + Cx2 * (unm1[idx + 1] - 2 * unm1[idx] + unm1[idx - 1])
                 + Cy2 * (unm1[idx + Nx] - 2 * unm1[idx] + unm1[idx - Nx])
                 + dt2 * f(i, j, t);
    }
}

int main() {
    double dx = a / Nx, dy = b / Ny;
    double Cx2 = Cx * Cx, Cy2 = Cy * Cy, c=1.2;
    double dt = Cx * dx/c, dt2 = dt * dt;
    int Nt = floor(T / dt);

    // Create events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    // double *x, *y;
    double *u, *unm1, *unm2;
    double *h_u;

    h_u = (double*)malloc(Nx*Ny*sizeof(double));

    int index=0;

    // We have to define the number of files in a smart way, since it may not result in a integer value
    int print_interval = ceil(T/dt/frames), print_counter=print_interval;
    int n_files = floor(T/dt/print_interval)+2;

    // Allocate memory on the host
    // hipMallocManaged(&x, Nx * sizeof(double));
    // hipMallocManaged(&y, Ny * sizeof(double));
    hipMallocManaged(&u, Nx * Ny * sizeof(double));
    hipMallocManaged(&unm1, Nx * Ny * sizeof(double));
    hipMallocManaged(&unm2, Nx * Ny * sizeof(double));

    // for (int i = 0; i < Nx; i++) x[i] = i * dx;
    // for (int j = 0; j < Ny; j++) y[j] = j * dy;

    dim3 blockSize(16, 16);
    dim3 gridSize((Nx + blockSize.x - 1) / blockSize.x, (Ny + blockSize.y - 1) / blockSize.y);

    step_zero_kernel<<<gridSize, blockSize>>>(unm2);

    step_one_kernel<<<gridSize, blockSize>>>(unm1, unm2, Cx2, Cy2, dt2);

    for (int t = 0; t < Nt; t++) {
        next_step_kernel<<<gridSize, blockSize>>>(u, unm1, unm2, t * dt, Cx2, Cy2, dt2);

        if(print_counter >= print_interval){
            hipMemcpy(h_u, u, Nx*Ny*sizeof(double), hipMemcpyDeviceToHost);
            write_linear_dir_matrix(h_u, Nx, Ny, "output/multi_matrix/", &index);
            print_counter = 0;
        }
        print_counter++;

        // Swap pointers
        double* temp = unm2;
        unm2 = unm1;
        unm1 = u;
        u = temp;
    }

    // Record the stop event
    hipEventRecord(stop, 0);

    // Wait for the events to finish
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    FILE *time_file;
    time_file = fopen("output/Execution_time_cuda.txt", "w");
    fprintf(time_file, "Elapsed Time:%lf\ndx = %g \t dy = %g\nNx = %d \t Ny = %d\na=%lf\nb=%lf\nT=%d\nc=%lf\nCx=%lf \t Cy=%lf",
    milliseconds/1000, dx, dy, Nx, Ny, a, b, T, c, Cx, Cy);

    // hipFree(x);
    // hipFree(y);
    hipFree(u);
    hipFree(unm1);
    hipFree(unm2);

    return 0;
}
